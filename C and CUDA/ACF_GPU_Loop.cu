#include "hip/hip_runtime.h"
#include <cstdio>
#include <cinttypes>
#include <chrono>
#include <bitset>


#define SIGNAL_LENGTH_MIN 5
#define SIGNAL_LENGTH_MAX 64


__global__ void calculate_acf(uint64_t start_offset, uint64_t end, uint64_t *min_result_sidelobe_amp, uint64_t *result_signal, uint64_t n) {
    
    // Variable initialization
    __shared__ int8_t acf_container[SIGNAL_LENGTH_MAX];
    __shared__ uint8_t signal_binary[SIGNAL_LENGTH_MAX];
    __shared__ uint64_t result_sidelobe_amp;

    // Shift by signal
    size_t idx = threadIdx.x; 

    // Decimal Signal
    uint64_t signal_decimal = blockIdx.x + start_offset; 

    while (signal_decimal <= end) {
        
        // Resetting the amplitude and splitting the signal into bits
        result_sidelobe_amp = 0;
        signal_binary[n - idx - 1] = (signal_decimal >> idx) & 1; //0 -> -1
        __syncthreads();

        // Start of ACF calculation for each shift
        int8_t temp = 0;
   
        //We turn the signal from the form {0;1} into {-1;1} and calculate the ACF
        for (size_t i = 0; i < n - idx; i++)  temp += (signal_binary[i + idx]*2-1) * (signal_binary[i]*2-1);
        
  
        // Taking the ACF modulo
		    if(temp<0) temp*=(-1);
        acf_container[idx] = temp;

        //Sidelobe amplitude calculation
        if (idx != 0) atomicMax(reinterpret_cast<unsigned long long int*>(&result_sidelobe_amp), (unsigned long long)acf_container[idx]);
        __syncthreads();

        // Checking if the available sidesheet amplitude is the best
        if (idx == 0) {
          uint64_t old = atomicMin(reinterpret_cast<unsigned long long int*>(min_result_sidelobe_amp), (unsigned long long)result_sidelobe_amp);
          if (old >= result_sidelobe_amp) {
              *result_signal = signal_decimal;
          }
        }
        signal_decimal += gridDim.x;
        __syncthreads();
    }
}


int is_goodlen(int n){
    if(n<SIGNAL_LENGTH_MIN || n>=SIGNAL_LENGTH_MAX){ 
        printf("Wrong signal length!"); 
        return 0;
    } 
    return 1;
}


uint64_t get_start_byblen(int n) {
	return 1ULL << (n - 1);
}


uint64_t get_end_byblen(int n) {
	return (1ULL << n) - 1ULL;
}

int main() {
    
    //Creation and initial design of the .dat file
    ofstream result_file;
    result_file.open("signal_acf_calculation_results.dat",  std::ios_base::app);
    result_file<<"Signal Length"<<TM_SPACES<<"Signal"<<TM_SPACES<<"Signal Dec"<<TM_SPACES<<"ACF"<<TM_SPACES<<"Calculation time (mks)"<<'\n';

    int n;
    result_file.close();
    for(n=SIGNAL_LENGTH_MIN; n<SIGNAL_LENGTH_MAX; n++){
      result_file.open("signal_acf_calculation_results.dat",  std::ios_base::app);
      auto start_time = std::chrono::high_resolution_clock::now();
       if(n<10) result_file<<"       ";
       else result_file<<"      ";
       result_file<<n<<"               ";

        //Variable initialization
        uint64_t *gpu_temporary_sidelobe_amp;
        uint64_t *gpu_temporary_result_signal;
        uint64_t end = get_end_byblen(n);
        uint64_t start = get_start_byblen(n);
        uint64_t result_sidelobe_amp;
        uint64_t result_signal;

        //Allocate memory
        hipMalloc((void**)&gpu_temporary_sidelobe_amp, sizeof(uint64_t));
        hipMalloc((void**)&gpu_temporary_result_signal, sizeof(uint64_t));
        hipMemset(gpu_temporary_sidelobe_amp, 0xFF, sizeof(uint64_t));

        //We calculate the best ACF for a signal of length n   
        calculate_acf<<<3071, n>>>(start, end, gpu_temporary_sidelobe_amp, gpu_temporary_result_signal, n);

        //We transfer data from the GPU to the CPU for further work
        hipMemcpy(&result_sidelobe_amp, gpu_temporary_sidelobe_amp, sizeof(uint64_t), hipMemcpyDeviceToHost);
        hipMemcpy(&result_signal, gpu_temporary_result_signal, sizeof(uint64_t), hipMemcpyDeviceToHost);

        //Preparing data for output and output itself
        std::bitset<SIGNAL_LENGTH_MAX> s(result_signal);
        auto end_time = std::chrono::high_resolution_clock::now();
        result_file<<s.to_string().c_str()<<"               "<<result_signal<<"                                             "<<result_sidelobe_amp<<"                                                  "<<(double)(std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count())<<'\n';
        result_file.close();
    }
    
    printf("Calculations done!");
    return 0;
}