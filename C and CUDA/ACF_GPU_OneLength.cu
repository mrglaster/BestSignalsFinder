// Works only with signals with length N. Calculation results'll be saved to file 'calc_results1.txt'

#include <hip/hip_runtime.h>
#include <cstdio>
#include <cinttypes>
#include <chrono>
#include <bitset>


#define SIGNAL_LENGTH_MIN 5
#define SIGNAL_LENGTH_MAX 64

/** Function calculating what we need*/
__global__ void calculate_acf(uint64_t start_offset, uint64_t end, uint64_t *min_result_sidelobe_amp, uint64_t *result_signal, uint64_t n) {
    
    // Variable initialization
    __shared__ int8_t acf_container[SIGNAL_LENGTH_MAX];
    __shared__ uint8_t signal_binary[SIGNAL_LENGTH_MAX];
    __shared__ uint64_t result_sidelobe_amp;

    // Shift by signal
    size_t idx = threadIdx.x; 

    // Decimal Signal
    uint64_t signal_decimal = blockIdx.x + start_offset; 

    while (signal_decimal <= end) {
        
        // Resetting the amplitude and splitting the signal into bits
        result_sidelobe_amp = 0;
        signal_binary[n - idx - 1] = (signal_decimal >> idx) & 1; //0 -> -1
        __syncthreads();

        // Start of ACF calculation for each shift
        int8_t temp = 0;
   
        //We turn the signal from the form {0;1} into {-1;1} and calculate the ACF
        for (size_t i = 0; i < n - idx; i++)  temp += (signal_binary[i + idx]*2-1) * (signal_binary[i]*2-1);
        
  
        // Taking the ACF modulo
		    if(temp<0) temp*=(-1);
        acf_container[idx] = temp;

        //Sidelobe amplitude calculation
        if (idx != 0) atomicMax(reinterpret_cast<unsigned long long int*>(&result_sidelobe_amp), (unsigned long long)acf_container[idx]);
        __syncthreads();

        // Checking if the available sidesheet amplitude is the best
        if (idx == 0) {
          uint64_t old = atomicMin(reinterpret_cast<unsigned long long int*>(min_result_sidelobe_amp), (unsigned long long)result_sidelobe_amp);
          if (old >= result_sidelobe_amp) {
              *result_signal = signal_decimal;
          }
        }
        signal_decimal += gridDim.x;
        __syncthreads();
    }
}

/**Does the signal's length correspond our requirements*/
int is_goodlen(int n){
    if(n<SIGNAL_LENGTH_MIN || n>=SIGNAL_LENGTH_MAX){ 
        printf("Wrong signal length!"); 
        return 0;
    } 
    return 1;
}

/**Get start by binary length*/
uint64_t get_start_byblen(int n) {
	return 1ULL << (n - 1);
}

/**Get end by binary length*/
uint64_t get_end_byblen(int n) {
	return (1ULL << n) - 1ULL;
}

/**The main function*/
int main() {
    
    //Signal Length. 
    int n = 15;

    //Signal length check
    if(!is_goodlen(n)) return -1;
  
    //The countdown has begun
    auto start_time = std::chrono::high_resolution_clock::now();

    //Variable initialization
    uint64_t *gpu_temporary_sidelobe_amp;
    uint64_t *gpu_temporary_result_signal;
    uint64_t end = get_end_byblen(n);
    uint64_t start = get_start_byblen(n);
    uint64_t result_sidelobe_amp;
    uint64_t result_signal;

    //Allocate memory
    hipMalloc((void**)&gpu_temporary_sidelobe_amp, sizeof(uint64_t));
    hipMalloc((void**)&gpu_temporary_result_signal, sizeof(uint64_t));
    hipMemset(gpu_temporary_sidelobe_amp, 0xFF, sizeof(uint64_t));

    //We calculate the best ACF for a signal of length N 
    calculate_acf<<<3072, n>>>(start, end, gpu_temporary_sidelobe_amp, gpu_temporary_result_signal, n);

    //We transfer data from the GPU to the CPU for further work
    hipMemcpy(&result_sidelobe_amp, gpu_temporary_sidelobe_amp, sizeof(uint64_t), hipMemcpyDeviceToHost);
    hipMemcpy(&result_signal, gpu_temporary_result_signal, sizeof(uint64_t), hipMemcpyDeviceToHost);

    //Preparing for the withdrawal and the conclusion itself
    std::bitset<SIGNAL_LENGTH_MAX> s(result_signal);
    printf("Best signal is %s (%llu) with result_sidelobe_amp of %llu\n", s.to_string().c_str(), result_signal, result_sidelobe_amp);

    //Timing completed
    auto end_time = std::chrono::high_resolution_clock::now();
    //Time is in microseconds
    printf("Calculation took %f nanoseconds\n", (double)(std::chrono::duration_cast<std::chrono::microseconds>(end_time - start_time).count()) );
    return 0;
}
